
#include <hip/hip_runtime.h>
#ifdef GPU

extern "C" {
#include <math.h>
#include <cuda.h>
#include <cuda_runtime_api.h>
#include <math.h>
#include "common.h"
}

#ifdef USE_SHARED
#include <cooperative_groups.h>
namespace cg = cooperative_groups;

template<class T>
struct SharedMemory
{
    __device__ inline operator T* ()
    {
        extern __shared__ int __smem[];
        return (T*)__smem;
    }

    __device__ inline operator const T* () const
    {
        extern __shared__ int __smem[];
        return (T*)__smem;
    }
};
#endif

#define BLOCK_SIZE 32

double4 *pos_dev = NULL;
double3 *acc_dev = NULL;
double* pos_host = NULL;
int inited = 0;
int N_store = 0;
int blockSize = BLOCK_SIZE;
int numBlocks = 0;

#ifdef USE_SHARED
int sharedMemSize = 0;
#define EPSILON 1e-200

__constant__ double softeningSquared_fp64;

cudaError_t setSofteningSquared(double softeningSq)
{
    return cudaMemcpyToSymbol(softeningSquared_fp64,
                              &softeningSq,
                              sizeof(double), 0,
                              cudaMemcpyHostToDevice);
};

__device__ double getSofteningSquared()
{
    return softeningSquared_fp64;
}

__device__ double3 bodyBodyInteraction(double3 ai, double4 bi, double4 bj)
{
    double3 r;

    // r_ij  [3 FLOPS]
    r.x = bj.x - bi.x;
    r.y = bj.y - bi.y;
    r.z = bj.z - bi.z;

    // distSqr = dot(r_ij, r_ij) + EPS^2  [6 FLOPS]
    double distSqr = r.x * r.x + r.y * r.y + r.z * r.z;

    // Softening is so small that it only impacts the case when bj = bi
    // In that case the evaluates acceleration is 0, as bj - bj = 0
    distSqr += getSofteningSquared();

    // invDistCube =1/distSqr^(3/2)  [4 FLOPS (2 mul, 1 sqrt, 1 inv)]
    double invDist = rsqrt(distSqr);
    double invDistCube = invDist * invDist * invDist;

    // s = m_j * invDistCube [1 FLOP]
    double s = bj.w * invDistCube;

    // a_i =  a_i + s * r_ij [6 FLOPS]
    ai.x += r.x * s;
    ai.y += r.y * s;
    ai.z += r.z * s;

    return ai;
}

__device__ double3 computeBodyAccel(double4 bodyPos,
                                    double4* positions,
                                    int numTiles, 
                                    cg::thread_block cta)
{
    double4* sharedPos = SharedMemory<double4>();

    double3 acc = { 0.0f, 0.0f, 0.0f };

    for (int tile = 0; tile < numTiles; tile++)
    {
        sharedPos[threadIdx.x] = positions[tile * blockDim.x + threadIdx.x];

        cg::sync(cta);

#pragma unroll BLOCK_SIZE
        for (unsigned int counter = 0; counter < blockDim.x; counter++)
        {
            acc = bodyBodyInteraction(acc, bodyPos, sharedPos[counter]);
        }
        cg::sync(cta);
    }
    return acc;
}

__global__ void gpuforce_shared(double4* __restrict__ p, int n, double3* __restrict__ acc, int numTiles) {
    cg::thread_block cta = cg::this_thread_block();
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Need to execute, even if i>=n as shared memory needs to be fully initialised
    double3 accel = computeBodyAccel(p[i], p, numTiles, cta);
    if (i < n)
    {
        acc[i].x = accel.x;
        acc[i].y = accel.y;
        acc[i].z = accel.z;
    }
}

#else
__global__ void gpuforce(double4* __restrict__ p, int n, double3* __restrict__ acc) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        double3 F = { 0.0f, 0.0f, 0.0f };

 //#pragma unroll
        for (int j = 0; j < n; j++) {
            double m = p[j].w;
            if (i == j || m == 0) continue;
            double dx = p[i].x - p[j].x;
            double dy = p[i].y - p[j].y;
            double dz = p[i].z - p[j].z;
            double distSqr = dx * dx + dy * dy + dz * dz;
            double invDist = rsqrt(distSqr);
            double invDist3 = invDist * invDist * invDist;

            F.x -= (m * dx * invDist3);
            F.y -= (m * dy * invDist3);
            F.z -= (m * dz * invDist3);
        }
        acc[i].x = F.x;
        acc[i].y = F.y;
        acc[i].z = F.z;
    }
}
#endif
extern "C" {
    void gpu_init(int N) {
        if (inited && N==N_store) return;
        // Clean up anything used previously
        printf("  gpu_init N=%d  ", N);
        gpu_finalize();

        numBlocks = ((int)N + blockSize - 1) / blockSize;

#ifdef USE_SHARED
        // Create a new data set - need to round up the size
        // of the pos block to multiple of block size and zero it (so mass is 0)
        int pos_size = numBlocks * blockSize;
        sharedMemSize = blockSize * sizeof(double4); // 4 doubles for pos
#else
        int pos_size = N;
#endif

        int err = 0;
        err = cudaMalloc(&pos_dev, pos_size * sizeof(double4));
        if (err > 0) {printf("cudaMalloc err = %d\n", err); exit(0); }
        err = cudaMemset(pos_dev, 0, pos_size * sizeof(double4));
        if (err > 0) { printf("cudaMemset err = %d\n", err); exit(0); }
        err = cudaMalloc(&acc_dev, N * sizeof(double3));
        if (err > 0) {printf("cudaMalloc err = %d\n", err); exit(0); }
        pos_host = (double*)malloc(N * 4 * sizeof(double));
        if (err > 0) { printf("cudaMalloc err = %d\n", err); exit(0); }

        inited = 1;
        N_store = N;

#ifdef USE_SHARED
        err = setSofteningSquared(EPSILON);
        if (err > 0) { printf("setSofteningSquared err = %d\n", err); exit(0); }
        printf("...GPU force SHARED opened. ");
#else
        printf("...GPU force opened. ");
#endif
        printf("blocksize = %d\n", blockSize);
    }

    void gpu_finalize() {
        if (pos_host)
            printf("Closing GPU force...\n");
        if (pos_dev != NULL) cudaFree(pos_dev);
        if (acc_dev != NULL) cudaFree(acc_dev);
        free(pos_host);
        pos_dev = NULL;
        acc_dev = NULL;
        pos_host = NULL;
        inited = 0;
        N_store = 0;
    }

    size_t ode_n_body_second_order_gpu(const real vec[], size_t N, real G, const real masses[], const real radii[], real acc[]) {
        if (masses == NULL) {printf("masses=NULL, exiting...\n"); exit(0);}

        cudaError_t err;
        gpu_init((int)N);

        for (size_t i = 0; i < N; i++) {
            pos_host[4 * i] = vec[3 * i];
            pos_host[4 * i + 1] = vec[3 * i + 1];
            pos_host[4 * i + 2] = vec[3 * i + 2];
            pos_host[4 * i + 3] = masses[i] * G;
        }

        err = cudaMemcpy(pos_dev, pos_host, N*sizeof(double4), cudaMemcpyHostToDevice);
        if (err > 0) {printf("cudaMemcpy err = %d, host_to_dev\n", err); exit(0); }

#ifdef USE_SHARED
        gpuforce_shared<<<numBlocks, blockSize, sharedMemSize >>>(pos_dev, (int)N, acc_dev, numBlocks);
#else   

        gpuforce<<<numBlocks, blockSize >>>(pos_dev, (int)N, acc_dev);
#endif
        err = cudaGetLastError();
        if (err != cudaSuccess) {printf("Error: %d %s\n", err, cudaGetErrorString(err)); exit(0);}


        // err = cudaMemcpy(acc_host, acc_dev, bytes, cudaMemcpyDeviceToHost);
        err = cudaMemcpy(acc, acc_dev, N*sizeof(double3), cudaMemcpyDeviceToHost);
        if (err != cudaSuccess) {printf("cudaMemcpy err = %d, %s\n", err, cudaGetErrorString(err)); exit(0); }

        return 0;
    }
} // end extern C

#endif
